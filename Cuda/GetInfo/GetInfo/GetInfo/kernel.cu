#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

using namespace std;




int main()
{
	int device;
	hipGetDevice(&device);

	struct hipDeviceProp_t props;

	hipGetDeviceProperties(&props, device);

	cout << "shared memory per block : " << props.name << endl;
	
	cout << "max thread per block: " << props.maxThreadsPerBlock << endl;
	cout << "max block per multiprocessor: " << props.maxBlocksPerMultiProcessor << endl;
	cout << "number of multiprocessor : " << props.multiProcessorCount << endl;
	cout << "max number of thread in total : " << props.multiProcessorCount * props.maxBlocksPerMultiProcessor * props.maxThreadsPerBlock << endl;

	cout << "max thread block dimension x : " << props.maxThreadsDim[0] << endl;
	cout << "max thread block dimension y : " << props.maxThreadsDim[1] << endl;
	cout << "max thread block dimension z : " << props.maxThreadsDim[2] << endl;
	cout << "max grid size x: " << props.maxGridSize[0] << endl;
	cout << "max grid size y: " << props.maxGridSize[1] << endl;
	cout << "max grid size z: " << props.maxGridSize[2] << endl;
	
	cout << "IPC support: " << props.ipcEventSupported << endl;

	cout << "shared memory per block : " << props.sharedMemPerBlock<< endl;
	cout << "shared memory per multiprocessor : " << props.sharedMemPerMultiprocessor << endl;

	cout << "L2 cache size: " << props.l2CacheSize << endl;
	cout << "persisting L2 cache max size: " << props.persistingL2CacheMaxSize << endl;
	cout << "access policy max window size: " << props.accessPolicyMaxWindowSize << endl;
	
	cout << "compute capability: " << props.computeMode << endl;

	


}


