#include "hip/hip_runtime.h"
// this version of the matrix multiplication use shared memory
#ifndef __HIPCC__  
#define __HIPCC__
#endif
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>

#define THREAD_BLOCK_SIZE 16

typedef struct {
	int width;
	int height;
	int stride;
	float* elements;
} Matrix;

__global__ void  MatMulKernel(const Matrix, const Matrix, Matrix);
void matMul(const Matrix A, const Matrix B, Matrix c);
void matrix_init(Matrix* M);

__device__ float GetElement(const Matrix A, int row, int col) {
	return A.elements[row * A.stride + col];
}

__device__ void SetElement(Matrix A, int row, int col, float value) {
	A.elements[row * A.stride + col] = value;
}

__device__ Matrix GetSubMatrix(Matrix A, int row, int col) {
	Matrix Asub;
	Asub.width = THREAD_BLOCK_SIZE;
	Asub.height = THREAD_BLOCK_SIZE;
	Asub.stride = A.stride;
	Asub.elements = &A.elements[A.stride * THREAD_BLOCK_SIZE * row + THREAD_BLOCK_SIZE * col];
	return Asub;
}



int main() {
	srand(time(NULL));

	Matrix A, B;
	A.width = 16;
	A.height = 16;
	B.width = 16;
	B.height = 16;
	matrix_init(&A);
	matrix_init(&B);

	Matrix C;
	C.width = 16;
	C.height = 16;
	C.elements = (float*)malloc(C.width * C.height * sizeof(float));

	matMul(A, B, C);

	for (size_t i = 0; i < C.height; i++)
	{
		for (size_t j = 0; j < C.width; j++)
		{
			std::cout << C.elements[i * C.width + j] << " ";
		}
		std::cout << std::endl;
	}
}
void matMul(const Matrix A, const Matrix B, Matrix C) {
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	d_A.stride = A.width;
	size_t size = A.width * A.height * sizeof(float);
	hipMalloc(&d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	d_B.stride = B.width;
	size = B.width * B.height * sizeof(float);
	hipMalloc(&d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;
	d_C.stride = C.width;
	size = C.width * C.height * sizeof(float);
	hipMalloc(&d_C.elements, size);

	//Invoke kernel
	dim3 dimBlock(THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE);
	dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
	MatMulKernel << <dimGrid, dimBlock >> > (d_A, d_B, d_C);

	//read result from device memory
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

	//free device memory
	hipFree(&d_A.elements);
	hipFree(&d_B.elements);
	hipFree(&d_C.elements);
}

__global__ void  MatMulKernel(const Matrix A, const Matrix B, Matrix C) {
	int blockRow = blockIdx.y;
	int blockCol = blockIdx.x;

	//each thread block computes one sub-matrix Csub of C
	Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

	//each thread accumulate the result in Cvalue
	float Cvalue = 0;

	//thread row and column
	int row = threadIdx.y;
	int col = threadIdx.x;

	for (size_t m = 0; m < (A.width/THREAD_BLOCK_SIZE); ++m)
	{
		//submatrix of A and B
		Matrix Asub = GetSubMatrix(A, blockRow, m);
		Matrix Bsub = GetSubMatrix(B, m, blockCol);

		//shared memory
		__shared__ float As[THREAD_BLOCK_SIZE][THREAD_BLOCK_SIZE];
		__shared__ float Bs[THREAD_BLOCK_SIZE][THREAD_BLOCK_SIZE];

		//each thread load one element of the sub matrix
		As[row][col] = GetElement(Asub, row, col);
		Bs[row][col] = GetElement(Bsub, row, col);

		//syncronize to make sure sub matrix are loaded
		__syncthreads();

		for (int e = 0; e < THREAD_BLOCK_SIZE; ++e)
		{
			Cvalue += As[row][e] * Bs[e][col];
		}
			
		__syncthreads();
		
	}
		SetElement(Csub, row, col, Cvalue);
}

void matrix_init(Matrix* M) {
	M->elements = (float*)malloc(M->width * M->height * sizeof(float));

	for (size_t i = 0; i < M->height; i++)
	{
		for (size_t j = 0; j < M->width; j++)
		{
			M->elements[i * M->width + j] = rand() % 100;
		}
	}
}
