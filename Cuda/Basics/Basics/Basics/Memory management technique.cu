#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <string.h>

__constant__ float constData[256];
__device__ float devData;
__device__ float devPointer;

void __constant__access();
__global__ void __constant__Access();

void __device__access();
__global__ void __device__Access();

void page_locked_memory();

int main() {
	//try page 28 code
	/*linear memory access */


	__constant__access();
	__device__access();
}

void __constant__access() {
	/*Constant memory is a global read-only cache which content can be broadcasted to multiple threads in a block.
	* every thread can access the data at the same time.
	* this variable reside in constant memory space and has the lifetime of the application
	* the referemce can be accessed from the host with hipGetSymbolAddress() and hipGetSymbolSize() but 
	* it is not possible to access the content of the pointer because it is in the device memory space
	*/

	float hostData[256];
	//populating hostData with progressive numbers
	for (size_t i = 0; i < 256; i++)
	{
		hostData[i] = i;
	}

	//copy hostData to constData
	hipMemcpyToSymbol(HIP_SYMBOL(constData), hostData, sizeof(hostData));

	//pupulating hostData with 0
	for (size_t i = 0; i < 256; i++)
	{
		hostData[i] = 0;
	}

	//every thread write the thread number in constData
	__constant__Access << < 16, 16 >> > ();

	//copy constData to hostData
	hipMemcpyFromSymbol(hostData, HIP_SYMBOL(constData), sizeof(hostData));
	std::cout << constData[10];

	size_t size;
	hipGetSymbolSize(&size, HIP_SYMBOL(constData));
	float* ptr;
	hipGetSymbolAddress((void**) & ptr, constData);

}

__global__ void __constant__Access() {
	int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	constData[tid] = tid;
}

void __device__access() {
	/*the __device__ variable and pointer exist only in the GPU global memory space*/
	/*do not mind the visual studio errors*/
	float value = 3.14f;
	hipMemcpyToSymbol(HIP_SYMBOL(devData), &value, sizeof(float));
	__device__Access<<<16, 16>>>();
	hipMemcpyFromSymbol(&value, HIP_SYMBOL(devData), sizeof(float));

	float* ptr;
	hipMalloc(&ptr, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(devPointer), &ptr, sizeof(ptr));
	
}

__global__ void __device__Access() {
	devData = 1234.56f;
}

void page_locked_memory() {
	/*page locked memory can be used by any device in the system
	* the driver tracks this portion of virtual memory and automatically accelerates copy functions
	* the memory pages wont change untill they exist in some CUDA context
	*/

	float* ptr;

	//The memory returned by this call will be considered as pinned memory by all CUDA contexts
	hipHostAlloc(&ptr, sizeof(float), hipHostMallocPortable);

	//Maps the allocation into the CUDA address space; this memory block has now 2 pointers.
	// hipHostAlloc() return host pointer, hipHostGetDevicePointer() return device pointer
	//since the memory are accessible from host and device they must be syncronized to avoid fail in critical section
	hipHostAlloc(&ptr, sizeof(float), hipHostMallocMapped);

	//Free the L1 and L2 cache after writing so this memory should be used only for writing because it makes reading worse
	hipHostAlloc(&ptr, sizeof(float), hipHostMallocWriteCombined);
}