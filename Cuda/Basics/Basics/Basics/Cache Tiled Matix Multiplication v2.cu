#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16

//matrix with metadata information
typedef struct {
    int width;
    int height;
    int stride;
    float* elements;
} Matrix;

// __device__ function can only be called from __global__ or __device__ functions
//get and set a matrix element value
__device__ float GetElement(const Matrix A, int row, int col) {
    return A.elements[row * A.stride + col];
}

__device__ void SetElement(const Matrix A, int row, int col, float value) {
    A.elements[row * A.stride + col] = value;
}

//submatrix with BLOCK_SIZE X BLOCK_SIZE dimension
__device__ Matrix GetSubMatrix(Matrix A, int row, int col) {
    Matrix Asub;
    Asub.width = BLOCK_SIZE;
    Asub.height = BLOCK_SIZE;
    Asub.stride = A.stride;
    Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row + BLOCK_SIZE * col]; //Pointer to the element of the Matrix where the subset start (the matrix in memory is linear)
    return Asub;
}

//actula gpu operations
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {

    // block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    //each threadblock compute one sumbmatrix
    Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

    float Cvalue = 0;

    //thread row and column withing Csub
    int row = threadIdx.y;
    int col = threadIdx.x;

    //loop over the entire matrix (note that non square matrix may need some adjustment for correct manipulation)
    for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {
        // A,B submatrix
        Matrix Asub = GetSubMatrix(A, blockRow, m);
        Matrix Bsub = GetSubMatrix(B, m, blockCol);

        //shered memory for A,B submatrix
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];    
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];  

        // each thread load one element of each submatrix
        As[row][col] = GetElement(Asub, row, col);
        Bs[row][col] = GetElement(Bsub, row, col);

        //make sure that all submatrix are loaded
        __syncthreads();

        for (int  e = 0; e < BLOCK_SIZE; ++e)
        {
            Cvalue += As[row][e] * Bs[e][col];
        }

        //make sure that  preceding computation is done before loading two new submatrix
        __syncthreads();
    }

    SetElement(Csub, row, col, Cvalue);
}

//host code for matrix multiplication
void MatMul(const Matrix A, const Matrix B, Matrix C) {
    //create and allocate Matrix in device memory
    Matrix d_A;
    d_A.width = d_A.stride = A.width;
    d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

    Matrix d_B;
    d_B.width = d_B.stride = B.width;
    d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

    Matrix d_C;
    d_C.width = d_C.stride = C.width;
    d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);

    //invoke kernel
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, B.height / dimBlock.y);
    MatMulKernel << <dimGrid, dimBlock >> > (d_A, d_B, d_C);

    //read C from device memory
    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

    //free memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}

Matrix InitMatrix(bool k) {
    int n = 1 << 10;
    Matrix K;
    K.height = 0;
    K.width = 0;
    K.stride = 0;
    K.elements = (float*)malloc(n * n * sizeof(float));

    if(k)
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            K.elements[i * n + j] = rand() % 100;
            
        }
    }

    return K;
}


int main()
{
    //1 initiate matrix with random numbers
    //0 only memory allocation
    Matrix A, B, C;
    A = InitMatrix(1);
    B = InitMatrix(1);
    C = InitMatrix(0);
    

    MatMul(A, B, C);

    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
