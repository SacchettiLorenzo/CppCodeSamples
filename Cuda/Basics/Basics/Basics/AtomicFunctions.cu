#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"

#include ""
#include "device_atomic_functions.h"



#include <stdio.h>



__global__ void addKernel(int *a)
{
    atomicAdd(a, 20);
    //atomicSub(a, 20);
    //atomicExch(a, 20);
    //atomicMin(a, 20);
    //atomicMax(a, 20);
    //atomicInc(a, 20);
    //atomicDec(a, 20);
    //atomicCas(a, 20);
    //atomicAnd(a, 20);
    //atomicOr(a, 20);
    //atomicXor(a, 20);

}

int main()
{
    //size = 1

    int A = 10;
    int B;

    //pointer to A
    int* cudaA = 0;

    // allocating and copying memory
    hipMalloc(&cudaA, sizeof(A));
    hipMemcpy(cudaA, &A, sizeof(A), hipMemcpyHostToDevice);


    addKernel << <1, 1 >> > (cudaA);

    hipMemcpy(&B, cudaA,sizeof(B), hipMemcpyDeviceToHost);

    printf("result %d\n", B);
    

    return 0;
}

