#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>


void vector_init(int* vector_ptr, int size);
__global__ void vectorAdd(int* a, int* b, int* c, int n);

int main() {

    int id = hipGetDevice(&id);

    int n = 1 << 16;

    size_t bytes = sizeof(int) * n;

    //unified memory pointers
    int* a, * b, * c;

    //allocate unified memory
    hipMallocManaged(&a, bytes);
    hipMallocManaged(&b, bytes);
    hipMallocManaged(&c, bytes);

    vector_init(a, n);
    vector_init(b, n);

    int NUM_THREAD = 256; // Arbitrary decided

    int NUM_BLOCKS = (int)ceil(n / NUM_THREAD);

    //transfer the data to the GPU asyncronously before their use
    hipMemPrefetchAsync(a, bytes, id);
    hipMemPrefetchAsync(b, bytes, id);

    vectorAdd << <NUM_BLOCKS, NUM_THREAD >> > (a, b, c, n);
     
    //wait for all the previos operations before using values
    hipDeviceSynchronize();
     
    //transfer the data back to the CPU asyncronously before their use
    hipMemPrefetchAsync(b, bytes, hipCpuDeviceId);

    for (size_t i = 0; i < n; i++)
    {
        std::cout << a[i] << " + " << b[i] << " = " << c[i] << std::endl;
    }
}

void vector_init(int* vector_ptr, int size) {
    for (size_t i = 0; i < size; i++)
    {
        vector_ptr[i] = rand() % 100;
    }
}

__global__ void vectorAdd(int* a, int* b, int* c, int n) {
    //Calculate the global thread ID (tid) used for the calculation as index

    int tid = (blockIdx.x * blockDim.x) + threadIdx.x; //using a specific block with fixed dimension and specific thread inside the block

    if (tid < n) {
        c[tid] = a[tid] + b[tid];
    }
}

/*
* This example does the same operation as the Vector Addition.cu 
* but use the unified memory to automatically manage the memory
* using cuda malloc management the data gets loaded into the gpu after the vectorAdd call causing a page fault
* the unified memory need a syncronization operation after performing operation on the gpu.
* 
* to perform the transfer before the vectorAdd call is possible to run a pre-feth operation asyncronously
* the same happen when using the data that shoud be transfered from the gpu to the cpu on call
* 
* hipMemPrefetchAsync send the data to the gpu/cpu before their actual use to avoid page fault
*/