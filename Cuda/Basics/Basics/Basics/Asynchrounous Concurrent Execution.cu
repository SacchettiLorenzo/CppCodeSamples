#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <string.h>

	__global__ void kernel(int*, int);
int main() {
	const int nStreams = 2;
	int streamSize = 64;
	int blockSize = 16;
	int* d_a;
	hipMalloc(&d_a, sizeof(int));
	int* a = (int*)malloc(sizeof(int));
	int streamBytes = 0;
	hipStream_t stream[nStreams];
	for (int i = 0; i < nStreams; i++)
	{
		hipStreamCreate(&stream[i]);
	}

	/*To implement the concurrent model, instead of calling hipMemcpy, 
	we call hipMemcpyAsync and launch kernel with the stream specified so that they will return to the host thread immediately after call.*/
	for (int i = 0; i < nStreams; i++)
	{
		int offset = i * streamSize;
		hipMemcpyAsync(&d_a[offset], &a[offset], sizeof(int), hipMemcpyHostToDevice, stream[i]);
		kernel << <streamSize / blockSize, blockSize, 0, stream[i] >> > (d_a, offset);
		hipMemcpyAsync(&a[offset], &d_a[offset], streamBytes, hipMemcpyDeviceToHost, stream[i]);
	}

	for (int i = 0; i < nStreams; i++)
	{
		hipStreamDestroy(stream[i]);
	}

	
}

__global__ void kernel(int* d_a, int) {
	//do somethig 
	*d_a = 10;
}
	/*some operation can be executed at the sime time
	* for example computation on the host, computation on the device and
	* memory tranfer can operate concurrently
	* concurrent operations are managed using streams that are a sequence of command to execute in order
	* 
	* one of the best way to use Async concurrent execution is to overlap data transfer and kernel execution
	* that could be usefull for the problems that can be solved with tiling
	* 
	* streams can be syncronized (even one by one) with the host
	*/

/* 
* continue host functions pag 50
*/