#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>

void vector_init(int* vector_ptr, int size);
__global__ void vectorAdd(int* a, int* b, int* c, int n);

int main()
{
    int n = 1 << 16;

    //host vector pointers
    int* h_a, * h_b, * h_c;

    //device vector pointers
    int* d_a, * d_b, * d_c;

    size_t bytes = sizeof(int) * n;

    //allocate host memory
    h_a = (int*)malloc(bytes);
    h_b = (int*)malloc(bytes);
    h_c = (int*)malloc(bytes);

    //allocate device memory
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    //initialize vectors with random values
    vector_init(h_a, n);
    vector_init(h_b, n);

    //comy memory from host to device
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    int NUM_THREAD = 256; // Arbitrary decided

    int NUM_BLOCKS = (int)ceil(n / NUM_THREAD);

    vectorAdd << <NUM_BLOCKS, NUM_THREAD >> > (d_a, d_b, d_c,n);

    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

    for (size_t i = 0; i < n; i++)
    {
        std::cout << h_a[i] << " + " << h_b[i] << " = " << h_c[i] << std::endl;
    }
}

void vector_init(int* vector_ptr, int size) {
    for (size_t i = 0; i < size; i++)
    {
        vector_ptr[i] = rand() % 100;
    }
}

__global__ void vectorAdd(int* a, int* b, int* c, int n) {
    //Calculate the global thread ID (tid) used for the calculation as index

    int tid = (blockIdx.x * blockDim.x) + threadIdx.x; //using a specific block with fixed dimension and specific thread inside the block

    if (tid < n) {
        c[tid] = a[tid] + b[tid];
    }
}


