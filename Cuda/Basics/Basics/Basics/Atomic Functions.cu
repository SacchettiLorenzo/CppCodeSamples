#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>

__constant__ __device__ int Dvalue = 10;

__constant__ __device__ unsigned int Dvalue_ = 20;
int value = 0;

__global__ void arithmetics(int);

__global__ void checkMemorySpace();


int main() {
	arithmetics << <1, 1 >> > (1);
	hipMemcpyFromSymbol(&value, HIP_SYMBOL(Dvalue), sizeof(int));
	std::cout << value << std::endl;
	arithmetics << <1, 1 >> > (2);
	hipMemcpyFromSymbol(&value, HIP_SYMBOL(Dvalue), sizeof(int));
	std::cout << value << std::endl;
	arithmetics << <1, 1 >> > (3);
	hipMemcpyFromSymbol(&value, HIP_SYMBOL(Dvalue), sizeof(int));
	std::cout << value << std::endl;
	arithmetics << <1, 1 >> > (4);
	hipMemcpyFromSymbol(&value, HIP_SYMBOL(Dvalue), sizeof(int));
	std::cout << value << std::endl;
	arithmetics << <1, 1 >> > (5);
	hipMemcpyFromSymbol(&value, HIP_SYMBOL(Dvalue_), sizeof(unsigned int));
	std::cout << value << std::endl;
	arithmetics << <1, 1 >> > (6);
	hipMemcpyFromSymbol(&value, HIP_SYMBOL(Dvalue_), sizeof(unsigned int));
	std::cout << value << std::endl;
	arithmetics << <1, 1 >> > (7);
	hipMemcpyFromSymbol(&value, HIP_SYMBOL(Dvalue), sizeof(int));
	std::cout << value << std::endl;
	arithmetics << <1, 1 >> > (8);
	hipMemcpyFromSymbol(&value, HIP_SYMBOL(Dvalue), sizeof(int));
	std::cout << value << std::endl;
	arithmetics << <1, 1 >> > (9);
	hipMemcpyFromSymbol(&value, HIP_SYMBOL(Dvalue), sizeof(int));
	std::cout << value << std::endl;
	arithmetics << <1, 1 >> > (10);
	hipMemcpyFromSymbol(&value, HIP_SYMBOL(Dvalue), sizeof(int));
	std::cout << value << std::endl;
	arithmetics << <1, 1 >> > (11);
	hipMemcpyFromSymbol(&value, HIP_SYMBOL(Dvalue), sizeof(int));
	std::cout << value << std::endl;
	checkMemorySpace << <1, 1 >> > ();
}

__global__ void arithmetics(int count) {
	
	switch (count)
	{
	case 1:
	atomicAdd(&Dvalue, 5);
		break;
	case 2:
		atomicSub(&Dvalue, 2);
		break;
	case 3:
		atomicMin(&Dvalue, 8);
		break;
	case 4:
		atomicMax(&Dvalue, 30);
		break;
	case 5:
		atomicInc(&Dvalue_, 30); //if Dvalue <= 30 -> Dvalue++
		break;
	case 6:
		atomicDec(&Dvalue_, 100); //if Dvalue <= 100 -> Dvalue--
		break;
	case 7:
		int compare = 30;
		atomicCAS(&Dvalue, compare, 11); //Compare And Swap:  if Dvalue == 30 -> Dvalue = 11;
		break;
	case 8:
		atomicExch(&Dvalue, 100); //exchange value
		break;
	case 9:
		atomicOr(&Dvalue, 200); 
		break;
	case 10:
		atomicAnd(&Dvalue, 1000); 
		break;
	case 11:
		atomicXor(&Dvalue, 500);
		break;
	default:
		break;
	}
}

__global__ void checkMemorySpace() {
	/*check the address in from a specific memory space*/
	__isGlobal(&Dvalue);
	__isShared(&Dvalue);
	__isConstant(&Dvalue);
	//__isGridConstant(&Dvalue);
	__isLocal(&Dvalue);
}