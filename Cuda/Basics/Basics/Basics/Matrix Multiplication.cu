#include "hip/hip_runtime.h"
// this version of the matrix multiplication does not use shared memory
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>

#define THREAD_BLOCK_SIZE 16

typedef struct {
	int width;
	int height;
	float* elements;
} Matrix;

__global__ void  MatMulKernel(const Matrix, const Matrix, Matrix);
void matMul(const Matrix A, const Matrix B, Matrix c);
void matrix_init(Matrix* M);

int main() {
	srand(time(NULL));

	Matrix A, B;
	A.width = 16;
	A.height = 16;
	B.width = 16;
	B.height = 16;
	matrix_init(&A);
	matrix_init(&B);

	Matrix C;
	C.width = 16;
	C.height = 16;
	C.elements = (float*)malloc(C.width * C.height * sizeof(float));

	matMul(A, B, C);

	for (size_t i = 0; i < C.height; i++)
	{
		for (size_t j = 0; j < C.width; j++)
		{
			std::cout << C.elements[i * C.width + j] << " ";
		}
		std::cout << std::endl;
	}
}

void matMul(const Matrix A, const Matrix B, Matrix C) {
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipMalloc(&d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	hipMalloc(&d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;
	size = C.width * C.height * sizeof(float);
	hipMalloc(&d_C.elements, size);

	//Invoke kernel
	dim3 dimBlock(THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE);
	dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
	MatMulKernel << <dimGrid, dimBlock >> > (d_A, d_B, d_C);

	//read result from device memory
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

	//free device memory
	hipFree(&d_A.elements);
	hipFree(&d_B.elements);
	hipFree(&d_C.elements);
}

__global__ void  MatMulKernel(const Matrix A, const Matrix B, Matrix C) {
	//each thread computes one element of C by accumulating result into Cvalue
	float Cvalue = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	for (size_t e = 0; e < A.width; ++e)
	{
		Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];
		C.elements[row * C.width + col] = Cvalue;
	}
}

void matrix_init(Matrix* M) {
	M->elements = (float*)malloc(M->width * M->height * sizeof(float));

	for (size_t i = 0; i < M->height; i++)
	{
		for (size_t j = 0; j < M->width; j++)
		{
			M->elements[i * M->width + j] = rand() % 100;
		}
	}
}
