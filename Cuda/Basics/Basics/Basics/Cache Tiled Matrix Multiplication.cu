#include "hip/hip_runtime.h"
#ifndef __HIPCC__  
#define __HIPCC__
#endif
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>

//shared memory with 16*16 size (int)
const int SHMEM_SIZE  = 1 << 10;
const int N = 1 << 10;

typedef struct {
	int width;
	int height;
	int stride;
	float* elements;
} Matrix;

__global__ void tiledMatrixMul(int* a, int* b, int* c, int n, int tile_size);
void matrix_init(int* matrix_ptr, int size);





int main() {
	//host matrix pointers
	int* h_a, * h_b, * h_c;

	//device matrix pointers
	int* d_a, * d_b, * d_c;

	size_t bytes = sizeof(int) * N * N;

	//allocate host memory
	h_a = (int*)malloc(bytes);
	h_b = (int*)malloc(bytes);
	h_c = (int*)malloc(bytes);

	//allocate device memory
	hipMalloc(&d_a, bytes);
	hipMalloc(&d_b, bytes);
	hipMalloc(&d_c, bytes);

	matrix_init(h_a, N*N);
	matrix_init(h_b, N*N);

	//comy memory from host to device
	hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

	int THREADS = 32;
	int BLOCKS = N / THREADS;

	dim3 threads(THREADS, THREADS);
	dim3 blocks(BLOCKS, BLOCKS);

	tiledMatrixMul << <blocks, threads >> > (d_a, d_b, d_c,N,16);

	hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
}

void matrix_init(int* matrix_ptr, int size) {
	for (size_t i = 0; i < size; i++)
	{
		matrix_ptr[i] = rand() % 100;
	}
}

__global__ void tiledMatrixMul(int* a, int* b, int* c, int n, int tile_size) {

	int temp = 0;

	//two staticallyt-sized pieces of shared memory
	__shared__ int A[SHMEM_SIZE];
	__shared__ int B[SHMEM_SIZE];

	//shorter parameters for clean re-use
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int bx = blockIdx.x;
	int by = blockIdx.y;

	//calculate global row and column position for this thread
	int row = by * tile_size + tx;
	int col = bx * tile_size + ty;

	for (size_t i = 0; i < (n / tile_size); i++)
	{
		A[(ty * tile_size) + tx] = a[row * n + (i * tile_size + tx)];
		B[(ty * tile_size) + tx] = b[(i * tile_size * n + ty * n) + col];

		//ensure that all thread have loaded their data before proceeding
		__syncthreads();

		for (size_t j = 0; j < tile_size; j++)
		{
			temp += A[(ty * tile_size) + j] * B[(j * tile_size) + tx];
		}

		//ensure that threads that have already finished computing start writing on memory
		__syncthreads();
	}

	c[(row * n) + col] = temp;
}