#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>

void allocation_1_Dimension();
void allocation_2_Dimension();
void allocation_3_Dimension();
__global__ void someFunction2D(int* mem_segment, size_t pitch, int width, int height);
__global__ void someFunction3D(hipPitchedPtr mem_segment, int width, int height, int depth);

int main() {
	//allocation_2_Dimension();
	allocation_3_Dimension();
}

void allocation_1_Dimension() {
	int size = 10;
	int* mem_segment;

	hipMalloc(&mem_segment, size * sizeof(int));

	for (size_t i = 0; i < size; i++)
	{
		mem_segment[i] = rand() % 100;
	}
}

void allocation_2_Dimension() {
	int width = 64; 
	int height = 64;

	int* mem_segment;
	size_t pitch;
	hipMallocPitch(&mem_segment, &pitch, width * sizeof(int), height);
	//std::cout <<"pitch: "<< pitch << std::endl;
	someFunction2D<< <100, 512 >> > (mem_segment, pitch, width, height);
}

void allocation_3_Dimension() {
	int width = 64;
	int height = 64;
	int depth = 64;

	hipExtent extent = make_hipExtent(width * sizeof(float),height, depth); //sizes of multidimensional array in linear memory
	hipPitchedPtr mem_segment; //3D memory pointer
	hipMalloc3D(&mem_segment, extent);
	someFunction3D << <100, 512 >> > (mem_segment, width, height,depth);
	
}

__global__ void someFunction2D(int* mem_segment, size_t pitch, int width, int height) {
	//accessig element in 2D array using given allignement
	for (size_t i = 0; i < height; i++)
	{
		int* row = (int*)((char*)mem_segment + i * pitch);
		for (size_t j = 0; j < width; ++j)
		{
			int element = row[j];
		}
	}
}

__global__ void someFunction3D(hipPitchedPtr mem_segment, int width, int height, int depth){
	//accessig element in 3D array using given allignement
	char* mem = (char*)mem_segment.ptr;
	size_t pitch = mem_segment.pitch;
	size_t slicePitch = pitch * height;
	for (size_t z = 0; z < depth; z++)
	{
		char* slice = mem + z + slicePitch; //2D array at index z
		for (size_t y = 0; y < height /*(mem_segment.ysize)*/; y++)
		{
			float* row = (float*)(slice + y * pitch); //1D array at index y
			for (size_t x = 0; x < width /*(mem_segment.xsize)*/; x++)
			{
				float element = row[x]; //elemet at index x
			}
		}
	}

}

/*
* hipMallocPitch() and hipMalloc3D() are reccomended for allocations of 2D and 3D array 
* as it make sure that the allocation in appropriatly padded to meet the allignement requirement
* the returned pith must be used to access array element
* the mem_segment type can be changed
* hipMallocPitch() make sure that the memory is allocated as a 2D array with (width * pitch) byte as width
* CudaMalloc3D() make sure that the memory is allocated as a 3D array, the dimesion are stored in a cudaExtend structure
* the overall size is given by width*sizeof(type). the ammount of byte for height and depth are calculated accordingly
* to access a 3D array we must slice the 3D structure in simpler 2D arrays then in rows.
*/